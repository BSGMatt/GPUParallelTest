#include "hip/hip_runtime.h"
//Running two versions of a function that adds two vectors together. 

#include <iostream>
#include <math.h>
#include <time.h>
#include "kernal.cuh"

/// Adds each element of a to each element of b. 
void add(int n, float* a, float* b) {
	for (int i = 0; i < n; i++) {
		b[i] = a[i] + b[i];
	}
}

//This code is run using the CPU. 
int addUsingCPU() {

	//keep track of how time the process took.
	clock_t start, end;
	start = clock();

	//Create two arrays that will be used as our vectors. 
	float* a = new float[N];
	float* b = new float[N];

	//Fill the two vectors
	for (uint32_t i = 0; i < N; i++) {
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	//add() will be the vector addition function
	add(N, a, b);

	end = clock();

	//Calculate the elapsed time
	double seconds = double(end - start) / double(1000000);

	//print the first 5 elements of b
	for (uint32_t i = 0; i < 5; i++) {
		printf("B[%d]: %f\n", i, b[i]);
	}

	printf("Process took %f seconds", seconds);

	//Free memory
	delete[] a;
	delete[] b;

	return 0;
}

int main() {

	printf("Adding with CPU: \n");

	addUsingCPU();

	printf("\n");

	addUsingGPU();
}