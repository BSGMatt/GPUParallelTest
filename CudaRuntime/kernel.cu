#include "hip/hip_runtime.h"
﻿//This program is designed to demonstrate the functionality of GPUPU by 
//Running two versions of a function that adds two vectors together. 

#include <iostream>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "kernal.cuh"


/// Adds each element of a to each element of b. 

__global__ void gpuAdd(int n, float* a, float* b) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		b[i] = a[i] + b[i];
	}
}

//This code is run using the GPU. 
int addUsingGPU() {

	//keep track of how time the process took.
	clock_t start, end;

	//Create two arrays containing the values that will be added together. 
	float* a, * b;
	hipError_t malloc_a = hipMallocManaged(&a, N * sizeof(float));
	hipError_t malloc_b = hipMallocManaged(&b, N * sizeof(float));

	//printf("hipError_t for a: %d, hipError_t for b: %d\n", malloc_a, malloc_b);

	for (uint32_t i = 0; i < N; i++) {
		a[i] = A;
		b[i] = B;
	}

	start = clock();

	//gpuAdd() will be the vector addition function
	gpuAdd<<<1, 1024>>>(N, a, b);
	//--//

	//printf("Last Error code: %d\n", hipGetLastError());

	hipDeviceSynchronize();

	end = clock();

	//Calculate the elapsed time
	double seconds = double(end - start) / double(10000000);

	//print the first 5 elements of b
	for (uint32_t i = 0; i < 5; i++) {
		printf("b[%d]: %.8f\n", i, b[i]);
	}


	printf("Process took %f seconds", seconds);

	//Free memory
	hipFree(a);
	hipFree(b);

	return 0;
}